#include "hip/hip_runtime.h"
#include <cstdio>
#include <iostream>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <cufftdx.hpp>

#include ""
#include "gpuTimer.h"
#include "matplotlibcpp.h"

#define cudaCheckError() {                                  \
    hipError_t e = hipGetLastError();                     \
    if (e != hipSuccess) {                                 \
        printf("CUDA error %s %d: %s\n",                    \
            __FILE__, __LINE__, hipGetErrorString(e));     \
        exit(EXIT_FAILURE);                                 \
    }                                                       \
}                                                           \

constexpr double pi = 3.141592655358979323846;

using namespace cufftdx;
namespace plt = matplotlibcpp;

void checkDeviceMemory(void)
{
    size_t free, total;
    hipMemGetInfo(&free, &total);
    auto free_mb = static_cast<float>(free)/1024.f/1024.f;
    auto total_mb = static_cast<float>(total)/1024.f/1024.f;
    std::cout << "Device memory (free/total) = " << free_mb << "/" << total_mb << " MB" << std::endl;
    std::cout << "Used memory = " << total_mb - free_mb << " MB" << std::endl;
}

int parse_option(int argc, char* argv[], const std::string& key) {
    for (int i = 1; i < argc - 1; ++i) {
        if (argv[i] == key) {
            try {
                return std::stoi(argv[i + 1]);
            }
            catch (const std::exception& e) {
                std::cerr << "Invalid value for " << key << ": " << argv[i + 1] << std::endl;
                std::exit(1);
            }
        }
    }
    
    std::cerr << "Invalid value for " << key << std::endl;
    std::exit(1);
}

void setGrid(
    std::vector<float>& x
    , std::vector<float>& k
    , float dx, float dk
    , const int N
    )
{
    for (auto i = 0; i < N; ++i) {
        x[i] = (i - N / 2) * dx;
        k[i] = (i - N / 2) * dk; 
    }
}

void generateGaussian(
    std::vector<float>& data
    , float x0 // shift
    , float dx
    , float sigma
    , float coeff = 1.f
    ) 
{
    int N = data.size();
    for (auto i = 0; i < N; ++i) {
        auto x = (i - N / 2) * dx;
        data[i] = coeff * expf(- (x - x0) * (x - x0) / (2.f * sigma * sigma));
    }
}

void generateAnswer(
    std::vector<float>& real  
    , std::vector<float>& imag   
    , float x0
    , float dk 
    , float sigma
    , float coeff
    )
{
    int N = real.size(); 
    for (auto i = 0; i < N; ++i) {
        auto k = (i - N / 2) * dk;
        auto envelope = coeff * expf(- 0.5f * k * k * sigma * sigma);
        real[i] = envelope * cosf(-k*x0);
        imag[i] = envelope * sinf(-k*x0);
    }
}

__global__ void toComplex(const float* data, hipfftComplex* out, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        out[i].x = data[i];
        out[i].y = 0.f;
    }
}

__global__ void toReal(const hipfftComplex* in, float* out, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        out[i] = in[i].x;
    }
}

template<typename T>
__global__ void fftshift(T* data, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N / 2) {
        T tmp = data[i];
        data[i] = data[i + N / 2];
        data[i + N / 2] = tmp;
    }
}

template<typename T>
__global__ void scale(T* input, T scale, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        input[i] = scale * input[i];
    }
}

template<>
__global__ void scale<hipfftComplex>(hipfftComplex* input, hipfftComplex scale, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        input[i] = hipCmulf(input[i], scale);
    }
}

// in-place
template<typename T>
void api_fftshift(T* input, const int size, hipStream_t stream) {
    int block_size = 128;
    int grid_shift = (size / 2 + block_size - 1) / block_size;
    fftshift<<<grid_shift, block_size, 0, stream>>>(input, size); // which one is faster? thrust? or this?
}

// in-place
template<typename T>
void api_scale(T* input, T scale_coeff, const int size, hipStream_t stream) {
    int block_size = 128;
    int grid_size = (size + block_size - 1) / block_size;
    scale<<<grid_size, block_size, 0, stream>>>(input, scale_coeff, size);
}

// out-of-place
void api_makeComplex(const float* in, hipfftComplex* out, const int size, hipStream_t stream) {
    int block_size = 128;
    int grid_size = (size + block_size - 1) / block_size;
    toComplex<<<grid_size, block_size, 0, stream>>>(in, out, size);
}

// out-of-place
void api_makeReal(const hipfftComplex* in, float* out, const int size, hipStream_t stream) {
    int block_size = 128;
    int grid_size = (size + block_size - 1) / block_size;
    toReal<<<grid_size, block_size, 0, stream>>>(in, out, size);
}
 
// out-of-place
void api_fftC2C(hipfftComplex* in, hipfftComplex* out, const int size, int direction, hipStream_t stream) {
    GpuTimer timer("api_fftC2C");
    timer.start();
    hipfftHandle plan;
    hipfftPlan1d(&plan, size, HIPFFT_C2C, 1);
    hipfftSetStream(plan, stream);
    hipfftExecC2C(plan, in, out, direction);
    hipfftDestroy(plan);

    hipStreamSynchronize(stream);
    timer.stop();
    timer.printElapsedTime();
} 

// out-of-place
void api_fftR2C(float* in, hipfftComplex* out, const int size, int direction, hipStream_t stream) {
    GpuTimer timer("api_fftR2C");
    timer.start();
    hipfftHandle plan;
    hipfftPlan1d(&plan, size, HIPFFT_R2C, 1);
    hipfftSetStream(plan, stream);
    hipfftExecR2C(plan, in, out); // R2C has no direction

    hipStreamSynchronize(stream);
    hipfftDestroy(plan);
    timer.stop();
    timer.printElapsedTime();
}

// out-of-place
void api_fftC2R(hipfftComplex* in, float* out, const int size, int direction, hipStream_t stream) {
    GpuTimer timer("api_fftC2R");
    timer.start();
    hipfftHandle plan;
    hipfftPlan1d(&plan, size, HIPFFT_C2R, 1);
    hipfftSetStream(plan, stream);
    hipfftExecC2R(plan, in, out); // C2R has no direction

    hipStreamSynchronize(stream);
    hipfftDestroy(plan);
    timer.stop();
    timer.printElapsedTime();
}

void fft_C2C(float* d_input
    , const int size
    , const float dx
    , int direction
    , hipStream_t stream
    , hipfftComplex* d_output
    )  
{
    hipfftComplex* d_cplx_input = nullptr;
    hipMallocAsync(&d_cplx_input, sizeof(hipfftComplex) * size, stream);

    // fftshift
    api_fftshift<float>(d_input, size, stream);
    
    // make complex
    api_makeComplex(d_input, d_cplx_input, size, stream);
    hipFreeAsync(d_input, stream);    
    
    // fft
    api_fftC2C(d_cplx_input, d_output, size, direction, stream);

    // free buffer
    hipFreeAsync(d_cplx_input, stream);    

    // fftshift
    api_fftshift<hipfftComplex>(d_output, size, stream); 

    // scale
    api_scale<hipfftComplex>(d_output, make_hipComplex(dx, 0.f), size, stream);
}

void fftdx_C2C(float* d_input
    , const int size
    , const float dx
    , int directdion
    , hipStream_t stream
    , hipfftComplex* d_output
    )
{
    using FFT = decltype(Size<size> 
        + Precision<float>()
        + Type<fft_type::c2c>()
        + Direction<fft_direction::forward>()
        + ElementsPerThread<8>() 
        + FFTsPerBlock<1>() 
        + SM<86>()
        + Block()
        );

    using complex_type = typename FFT::value_type;

    auto size = FFT::ffts_per_block * cufftdx::siuzeof<FFT>::value;
        
}

void ifft_C2C(hipfftComplex* d_input
    , const int size
    , const float dx
    , int direction
    , hipStream_t stream
    , float* d_output
    )
{
    hipfftComplex* d_cplx_output = nullptr; 
    hipMallocAsync(&d_cplx_output, sizeof(hipfftComplex) * size, stream);

    // fftshift  
    api_fftshift<hipfftComplex>(d_input, size, stream);
    
    // ifft
    api_fftC2C(d_input, d_cplx_output, size, direction, stream);

    // fftshift
    api_fftshift<hipfftComplex>(d_cplx_output, size, stream);

    // scale
    api_scale<hipfftComplex>(d_cplx_output, make_hipComplex(1.f/(dx * size), 0.f), size, stream);

    // to_real
    api_makeReal(d_cplx_output, d_output, size, stream);
}

void ifft_C2R(hipfftComplex* d_input
    , const int size
    , const float dx
    , int direction
    , hipStream_t stream
    , float* d_output
    )
{
    // ifft
    api_fftC2R(d_input, d_output, size, direction, stream);

    // fftshift
    api_fftshift<float>(d_output, size, stream);

    // scale
    api_scale<float>(d_output, 1.f/(dx * size), size, stream);
}

void fft_R2C(float* d_input
    , const int size
    , const float dx
    , int direction
    , hipStream_t stream    
    , hipfftComplex* d_output_r2c
    )  
{
    // fftshift
    api_fftshift<float>(d_input, size, stream);

    // fft r2c test
    api_fftR2C(d_input, d_output_r2c, size, direction, stream);

    // scale
    api_scale<hipfftComplex>(d_output_r2c, make_hipComplex(dx, 0.f), size / 2 + 1, stream);
}

void test_C2C(
    float* d_output
    , const int N
    , const float x0
    , const float dx
    , const float sigma
    , hipStream_t stream
    )
{
    std::vector<float> h_fx(N);
    hipMemcpyAsync(h_fx.data(), d_output, sizeof(float) * N, hipMemcpyDeviceToHost, stream); 

    std::vector<float> h_fx_theory(N);
    generateGaussian(h_fx_theory, x0, dx, sigma);

    for (auto i = 0u; i < N; ++i) {
        if (abs(h_fx[i] - h_fx_theory[i]) > 0.1) 
        {
            std::cout << "large numerical differences! test failed" << std::endl;
        }
    }
    std::cout << "accuracy test: success" << std::endl;
}

void test_R2C(
    float* d_output
    , const int N
    , const float x0
    , const float dx
    , const float sigma
    , hipStream_t stream
    )
{
    std::vector<float> h_fx(N);
    hipMemcpyAsync(h_fx.data(), d_output, sizeof(float) * N, hipMemcpyDeviceToHost, stream); 

    std::vector<float> h_fx_theory(N);
    generateGaussian(h_fx_theory, x0, dx, sigma);

    for (auto i = 0u; i < N; ++i) {
        if (abs(h_fx[i] - h_fx_theory[i]) > 0.1) 
        {
            std::cout << "large numerical differences! test failed" << std::endl;
        }
    }
    std::cout << "accuracy test: success" << std::endl;
}

int main(int argc, char* argv[])
{
    // data preparation 
    const int N = parse_option(argc, argv, "--width");
    const int mode = parse_option(argc, argv, "--mode");
    const float dx = 8.f;
    const float sigma = 100.f;
    const float width = dx * N;
    const float dk = 2.f * pi / (dx * N);
    const float x0 = 14.f * dx;

    auto h_x = std::vector<float>(N);
    auto h_k = std::vector<float>(N);
    setGrid(h_x, h_k, dx, dk, N);

    auto h_input = std::vector<float>(N);
    generateGaussian(h_input, x0, dx, sigma);

    // plot
    plt::plot(h_x, h_input);
    plt::show();
    
    // cuda stream
    hipStream_t stream;
    hipStreamCreate(&stream);

    // gpu memory allocation
    float* d_input = nullptr;
    float* d_input_check = nullptr;
    float* d_input_check_r2c = nullptr;
    hipfftComplex* d_output = nullptr;
    hipfftComplex* d_output_r2c = nullptr;

    hipMallocAsync(&d_input, sizeof(float) * N, stream);
    hipMallocAsync(&d_output, sizeof(hipfftComplex) * N, stream);
    hipMallocAsync(&d_output_r2c, sizeof(hipfftComplex) * (N/2 + 1), stream);
    hipMallocAsync(&d_input_check, sizeof(float) * N, stream);
    hipMallocAsync(&d_input_check_r2c, sizeof(float) * N, stream);
    hipMemcpyAsync(d_input, h_input.data(), sizeof(float) * N, hipMemcpyHostToDevice, stream);

    // for GPU warming up
    std::cout << "GPU warming up - start" << std::endl;
    api_fftR2C(d_input, d_output_r2c, N, HIPFFT_FORWARD, stream);
    std::cout << "GPU warming up - end" << std::endl;

    GpuTimer timer("main_algorithm_" + std::to_string(N) + "_" + std::to_string(mode));
    timer.start();

    switch (mode) {
        case 0:
            fft_C2C(d_input, N, dx, HIPFFT_FORWARD, stream, d_output);
            break;
        case 1:
            fft_R2C(d_input, N, dx, HIPFFT_FORWARD, stream, d_output_r2c);
            break;
        case 2:
            fftdx_C2C(d_input, N, dx, HIPFFT_FORWARD, stream, d_output);
        default:
            fft_C2C(d_input, N, dx, HIPFFT_FORWARD, stream, d_output); 
            break;
    }

    checkDeviceMemory();
    timer.stop();
    timer.printElapsedTime();

    hipStreamSynchronize(stream);
    
    switch(mode) {
        case 0:
            ifft_C2C(d_output, N, dx, HIPFFT_BACKWARD, stream, d_input_check);
            break;
        case 1:
            ifft_C2R(d_output_r2c, N, dx, HIPFFT_BACKWARD, stream, d_input_check_r2c);
            break;
        default:
            ifft_C2C(d_output, N, dx, HIPFFT_BACKWARD, stream, d_input_check);
            break;
    }

    switch(mode) {
        case 0:
            test_C2C(d_input_check, N, x0, dx, sigma, stream);
            break;
        case 1:
            test_R2C(d_input_check_r2c, N, x0, dx, sigma, stream);
            break;
        default:
            test_C2C(d_input_check, N, x0, dx,  sigma, stream);
            break;
    }

    std::cout << std::endl;
    
//    // plot
//    plt::plot(h_k, h_fk_real);
//    plt::show();

    // clean up
    hipFreeAsync(d_output, stream);    

    return 0;
}

