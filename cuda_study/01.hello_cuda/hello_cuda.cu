#include "hip/hip_runtime.h"

#include <stdio.h>

#define cudaCheckError() {                                  \
    hipError_t e = hipGetLastError();                     \
    if (e != hipSuccess) {                                 \
        printf("CUDA error %s %d: %s\n",                    \
            __FILE__, __LINE__, hipGetErrorString(e));     \
        exit(EXIT_FAILURE);                                 \
    }                                                       \
}                                                           \

void checkDeviceMemory(void)
{
    size_t free, total;
    hipMemGetInfo(&free, &total);
    printf("Device memory (free/total) = %lld/%lld bytes\n", free, total);
}

__global__ void helloCUDA(void)
{
    printf("Hello CUDA from GPU!\n");
}

int main(void)
{
    printf("Hello GPU from CPU!\n");
    helloCUDA<<<2, 100>>>();

    hipDeviceSynchronize();

    int* dDataPtr;
    hipError_t errorCode;

    checkDeviceMemory();
    errorCode = hipMalloc(&dDataPtr, sizeof(int) * 1024 * 1024);
    printf("hipMalloc - %s\n", hipGetErrorName(errorCode));
    checkDeviceMemory();

    hipDeviceSynchronize();
 
    return 0;
}

